#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <clocale>

hipError_t Armstrong(int *result, unsigned int size);

bool IsArmstrong(int N);

__global__ void Kernel(int *result, unsigned int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d \n", i);
	if (i >= size) return;
	int n = i+100;
	int a = n % 10;
	n /= 10;
	int b = n % 10;
	n /= 10;
	int c = n % 10;

	if (a*a*a + b * b*b + c * c*c == i+100)
	{
		result[i] = i+100;
		//printf(" number[%d] = %d \n", i, numbers[i]);
	}
}

int main()
{
	setlocale(LC_CTYPE, "rus");

	const int arraySize = 900;
	int result[arraySize] = { 0 };

	hipError_t cudaStatus = Armstrong(result, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}


	printf("��������� � GPU: \n");

	for (int i = 0; i<arraySize; i++)
	{
		if (result[i]!=0)
		{
			printf("%d \n", result[i]);
		}
	}

	printf("\n��������� � CPU: \n");
	for (int i=100; i<1000; i++)
	{
		if(IsArmstrong(i)) printf("%d \n", i);
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t Armstrong(int *result, unsigned int size)
{
	int *dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	
	cudaStatus = hipMalloc((void**)&dev_result, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy(dev_result, result, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 block(32, 1);
	dim3 grid((size / 32 + 1), 1);
	Kernel << <grid, block >> > (dev_result, size);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);

	return cudaStatus;
}

bool IsArmstrong(int N)
{
	int sum = 0;
	int n = N;

	while (n!=0)
	{
		int a = n % 10;
		sum += a * a*a;
		n /= 10;
	}
	return sum==N;
}
